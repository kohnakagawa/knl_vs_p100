#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <random>
#include <chrono>
#include "cuda_ptr.cuh"

__global__ void daxpy(const double* __restrict__ x,
                      const double* __restrict__ y,
                      double* __restrict__ z,
                      const double s,
                      const int val_size) {
  const auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < val_size) {
    z[tid] = s * x[tid] + y[tid];
  }
}

__global__ void daxpy2(const double2* __restrict__ x,
                       const double2* __restrict__ y,
                       double2* __restrict__ z,
                       const double s,
                       const int val_size) {
  const auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < val_size) {
    z[tid].x = s * x[tid].x + y[tid].x;
    z[tid].y = s * x[tid].y + y[tid].y;
  }
}

__attribute__((noinline))
void reference(const std::vector<double>& x,
               const std::vector<double>& y,
               std::vector<double>& z,
               const double s) {
  const int size = x.size();
  for (int i = 0; i < size; i++) {
    z[i] = s * x[i] + y[i];
  }
}

void check(const std::vector<double>& z_ref,
           const double* z) {
  const auto size = z_ref.size();
  for (size_t i = 0; i < size; i++) {
    if (z_ref[i] != z[i]) {
      std::cout << "mismatch\n";
      std::cout << i << " " << z_ref[i] << " " << z[i] << std::endl;
      std::exit(1);
    }
  }
}

#define BENCH(repr, size)                                               \
  do {                                                                  \
    using namespace std::chrono;                                        \
    const int LOOP = 10000;                                             \
    const auto beg = system_clock::now();                               \
    for (int i = 0; i < LOOP; i++) repr;                                \
    const auto end = system_clock::now();                               \
    const double dur = duration_cast<milliseconds>(end - beg).count();  \
    const double band_width =                                           \
      3.0 * size * sizeof(double) / ((dur * 1.0e-3  / double(LOOP)) * 1.0e9); \
    std::cerr << "array " << size << " " << band_width << " [GB/s] ";   \
    std::cerr << dur <<  " [ms]\n";                                     \
  } while(0)

#define BENCH_CUDA(repr, size, gl_size, tb_size, ...)                   \
  do {                                                                  \
    using namespace std::chrono;                                        \
    const int LOOP = 10000;                                             \
    const auto beg = system_clock::now();                               \
    for (int i = 0; i < LOOP; i++)                                      \
      repr<<<gl_size, tb_size>>>(__VA_ARGS__);                          \
    checkCudaErrors(hipDeviceSynchronize());                           \
    const auto end = system_clock::now();                               \
    const double dur = duration_cast<milliseconds>(end - beg).count();  \
    const double band_width =                                           \
      3.0 * size * sizeof(double) / ((dur * 1.0e-3  / double(LOOP)) * 1.0e9); \
    std::cerr << "array " << size << " " << band_width << " [GB/s] ";   \
    std::cerr << dur <<  " [ms]\n";                                     \
  } while(0)

int main(const int argc, const char* argv[]) {
  int val_size = 1 << 25;
  int tb_size = 128;
  if (argc >= 2) {
    val_size = std::atoi(argv[1]);
  }
  if (argc == 3) {
    tb_size = std::atoi(argv[2]);
  }

  if (tb_size < 64 || tb_size > 1024) {
    std::cerr << "thread block size is not appropriate.\n";
    std::exit(1);
  }

  const double s = 2.0;

  cuda_ptr<double> x_vec, y_vec, z_vec;
  cuda_ptr<double2> x2_vec, y2_vec, z2_vec;

  x_vec.allocate(val_size); x2_vec.allocate(val_size / 2);
  y_vec.allocate(val_size); y2_vec.allocate(val_size / 2);
  z_vec.allocate(val_size); z2_vec.allocate(val_size / 2);

  std::mt19937 mt;
  std::uniform_real_distribution<double> urd(0, 1.0);
  std::generate_n(&x_vec[0], val_size, [&mt, &urd](){return urd(mt);});
  std::generate_n(&y_vec[0], val_size, [&mt, &urd](){return urd(mt);});
  std::fill_n(&z_vec[0], val_size, 0.0);

  for (int i = 0; i < val_size; i += 2) {
    x2_vec[i / 2].x = x_vec[i    ];
    x2_vec[i / 2].y = x_vec[i + 1];
    y2_vec[i / 2].x = y_vec[i    ];
    y2_vec[i / 2].y = y_vec[i + 1];
  }

  x_vec.host2dev(); x2_vec.host2dev();
  y_vec.host2dev(); y2_vec.host2dev();
  z_vec.host2dev(); z2_vec.host2dev();

  std::vector<double> x_vec_ref(val_size), y_vec_ref(val_size), z_vec_ref(val_size);

  std::copy_n(&x_vec[0], val_size, x_vec_ref.begin());
  std::copy_n(&y_vec[0], val_size, y_vec_ref.begin());
  std::copy_n(&z_vec[0], val_size, z_vec_ref.begin());
  reference(x_vec_ref, y_vec_ref, z_vec_ref, s);


  auto gl_size = (val_size - 1) / tb_size + 1;
  // BENCH_CUDA(daxpy, val_size, gl_size, tb_size, x_vec, y_vec, z_vec, s, val_size);
  // z_vec.dev2host();
  // check(z_vec_ref, &z_vec[0]);

  gl_size = (val_size / 2 - 1) / tb_size + 1;
  BENCH_CUDA(daxpy2, val_size, gl_size, tb_size, x2_vec, y2_vec, z2_vec, s, val_size / 2);
  z2_vec.dev2host();
  check(z_vec_ref, &z2_vec[0].x);
}
