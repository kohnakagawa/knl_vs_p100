
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

__global__ void nop() {}

void func(const int loop = 10000) {
  using namespace std::chrono;

  const auto beg = system_clock::now();
  for (int i = 0; i < loop; i++) {
    nop<<<1000, 128>>>();
  }
  hipDeviceSynchronize();
  const auto end = system_clock::now();
  std::cout << static_cast<double>(duration_cast<microseconds>(end - beg).count()) / static_cast<double>(loop)
            << " [microseconds]\n";
}

int main() {
  func();
}
