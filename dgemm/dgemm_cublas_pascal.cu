#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <chrono>
#include <hipblas.h>
#include <cassert>
#include "cuda_ptr.cuh"

static inline void __cublasSafeCall(hipblasStatus_t err,
                                    const char *file,
                                    const int line) {
  if (HIPBLAS_STATUS_SUCCESS != err) {
    fprintf(stderr, "CUBLAS error in file '%s', line %d\n \nerror %d \nterminating!\n", file, line, err);
    hipDeviceReset();
    assert(0);
  }
}

#ifndef cublasSafeCall
#define cublasSafeCall(err)     __cublasSafeCall(err, __FILE__, __LINE__)
#endif

int mat_size = 10000;

void dgemm_cublas(const cuda_ptr<double>& x,
                  const cuda_ptr<double>& y,
                  cuda_ptr<double>& z,
                  const hipblasHandle_t& handle) {
  const double alpha = 1.0, beta = 0.0;
  cublasSafeCall(hipblasDgemm(handle,
                             HIPBLAS_OP_N, HIPBLAS_OP_N,
                             mat_size, mat_size, mat_size,
                             &alpha,
                             x, mat_size,
                             y, mat_size,
                             &beta,
                             z, mat_size));
}

void show_version(const hipblasHandle_t& handle) {
  int cublas_version = 0;
  cublasSafeCall(cublasGetVersion(handle, &cublas_version));
  std::cout << "cublas version " << cublas_version << "\n";
}

#define BENCH(repr)                                                     \
  do {                                                                  \
    using namespace std::chrono;                                        \
    const auto beg = system_clock::now();                               \
    repr;                                                               \
    checkCudaErrors(hipDeviceSynchronize());                           \
    const auto end = system_clock::now();                               \
    const double elapsed =                                              \
      1.0e-3 * duration_cast<milliseconds>(end - beg).count();          \
    const double flops =                                                \
      1.0e-9 * (2.0 * mat_size - 1) * mat_size * mat_size / elapsed;    \
    std::cerr << "array " << mat_size << " " << flops << " [GFLOPS] ";  \
    std::cerr << elapsed << " [ms]\n";                                  \
  } while (0)

int main(const int argc, const char* argv[]) {
  if (argc >= 2) mat_size = std::atoi(argv[1]);

  cuda_ptr<double> x_mat, y_mat, z_mat;

  x_mat.allocate(mat_size * mat_size);
  y_mat.allocate(mat_size * mat_size);
  z_mat.allocate(mat_size * mat_size);

  for (int i = 0; i < mat_size * mat_size; i++) {
    x_mat[i] = double(i + 1);
    y_mat[i] = double(- i - 1);
    z_mat[i] = 0.0;
  }

  x_mat.host2dev();
  y_mat.host2dev();
  z_mat.host2dev();

  hipblasHandle_t handle;
  cublasSafeCall(hipblasCreate(&handle));
  show_version(handle);
  BENCH(dgemm_cublas(x_mat, y_mat, z_mat, handle));
  cublasSafeCall(hipblasDestroy(handle));
  z_mat.dev2host();

  printf (" Top left corner of matrix x_mat: \n");
  for (int i = 0; i < std::min(mat_size, 6); i++) {
    for (int j = 0; j < std::min(mat_size, 6); j++) {
      printf ("%12.0f", x_mat[j + i * mat_size]);
    }
    printf ("\n");
  }

  printf ("\n Top left corner of matrix y_mat: \n");
  for (int i = 0; i < std::min(mat_size, 6); i++) {
    for (int j = 0; j < std::min(mat_size, 6); j++) {
      printf ("%12.0f", y_mat[j + i * mat_size]);
    }
    printf ("\n");
  }

  printf ("\n Top left corner of matrix z_mat: \n");
  for (int i = 0; i < std::min(mat_size, 6); i++) {
    for (int j = 0; j < std::min(mat_size, 6); j++) {
      printf ("%12.5G", z_mat[j + i * mat_size]);
    }
    printf ("\n");
  }
}
