#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <random>
#include <chrono>
#include <fstream>
#include "cuda_ptr.cuh"

#if __CUDACC_VER_MAJOR__ < 8
__device__ __forceinline__ double atomicAdd(double* address, double val) {
  auto address_as_ull = reinterpret_cast<unsigned long long int*>(address);
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}
#endif

typedef int Dtype;
// typedef double Dtype;

__global__ void make_hist(const int* val,
                          Dtype* bin,
                          const int val_size) {
  const auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < val_size) {
    const auto dst = val[tid];
    atomicAdd(&bin[dst], 1);
  }
}

__attribute__((noinline))
void reference(const std::vector<int>& val,
               std::vector<Dtype>& bin) {
  for (const auto v : val) {
    bin[v] += 1.0;
  }
}

void check(const std::vector<Dtype>& bin_ref,
           const cuda_ptr<Dtype>& bin) {
  const auto size = bin_ref.size();
  for (size_t i = 0; i < size; i++) {
    if (bin_ref[i] != bin[i]) {
      std::cout << "mismatch\n";
      std::cout << bin_ref[i] << " " << bin[i] << std::endl;
      std::exit(1);
    }
  }
}

#define BENCH(repr, max, size, sd)                                      \
  do {                                                                  \
    using namespace std::chrono;                                        \
    const int LOOP = 100;                                               \
    const auto beg = system_clock::now();                               \
    for (int i = 0; i < LOOP; i++) repr;                                \
    const auto end = system_clock::now();                               \
    const double dur = duration_cast<milliseconds>(end - beg).count();  \
    std::cerr << "range [0 : " << max << "] ";                          \
    if (sd == 0) {                                                      \
      std::cerr << "uniform ";                                          \
    } else {                                                            \
      std::cerr << "sd " << sd << " ";                                  \
    }                                                                   \
    std::cerr << "array size " << size << " ";                          \
    std::cerr << dur << " [ms]\n";                                      \
  } while(0)

#define BENCH_CUDA(repr, max, size, sd, gr_size, tb_size, ...)          \
  do {                                                                  \
    using namespace std::chrono;                                        \
    const int LOOP = 100;                                               \
    const auto beg = system_clock::now();                               \
    for (int i = 0; i < LOOP; i++)                                      \
      repr<<<gr_size, tb_size>>>(__VA_ARGS__) ;                         \
    checkCudaErrors(hipDeviceSynchronize());                           \
    const auto end = system_clock::now();                               \
    const double dur = duration_cast<milliseconds>(end - beg).count();  \
    std::cerr << "range [0 : " << max << "] ";                          \
    if (sd == 0) {                                                      \
      std::cerr << "uniform ";                                          \
    } else {                                                            \
      std::cerr << "sd " << sd << " ";                                  \
    }                                                                   \
    std::cerr << "array size " << size << " ";                          \
    std::cerr << dur << " [ms]\n";                                      \
  } while (0)

int main(const int argc, const char* argv[]) {
  int val_size = 10000000;
  int bin_size = 1000;
  double sd = 0;
  if (argc >= 2) val_size = std::atoi(argv[1]);
  if (argc >= 3) bin_size = std::atoi(argv[2]);
  if (argc >= 4) sd       = std::atof(argv[3]);

  cuda_ptr<int> val;
  cuda_ptr<Dtype> bin;

  val.allocate(val_size);
  bin.allocate(bin_size);

  std::mt19937 mt;
  if (sd == 0) {
    std::uniform_int_distribution<> uid(0, bin_size - 1);
    std::generate_n(&val[0], val_size, [&mt, &uid](){return uid(mt);});
  } else if (sd > 0.0) {
    std::normal_distribution<> nd(bin_size / 2, sd);
    int cnt = 0;
    while (true) {
      const auto ret = int(std::floor(nd(mt)));
      if (ret >= 0 && ret < bin_size) val[cnt++] = ret;
      if (cnt == val_size) break;
    }
  } else {
    std::cerr << "sd should be >= 0.\n";
    std::exit(1);
  }

  std::fill_n(&bin[0], bin_size, 0.0);

  val.host2dev();
  bin.host2dev();

  std::vector<int> val_ref(val_size);
  std::vector<Dtype> bin_ref(bin_size);

  std::copy_n(&val[0], val_size, val_ref.begin());
  BENCH(reference(val_ref, bin_ref), bin_size, val_size, sd);

  const auto tb_size = 128;
  const auto gr_size = (val_size - 1) / tb_size + 1;
  BENCH_CUDA(make_hist, bin_size, val_size, sd, gr_size, tb_size, val, bin, val_size);

  bin.dev2host();
  check(bin_ref, bin);
}
