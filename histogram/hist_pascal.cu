#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <random>
#include <chrono>
#include "cuda_ptr.cuh"

#if __CUDACC_VER_MAJOR__ < 8
__device__ __forceinline__ double atomicAdd(double* address, double val) {
  auto address_as_ull = reinterpret_cast<unsigned long long int*>(address);
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}
#endif

__global__ void make_hist(const int* val,
                          double* bin,
                          const int val_size) {
  const auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < val_size) {
    const auto dst = val[tid];
    atomicAdd(&bin[dst], 1.0);
  }
}

__attribute__((noinline))
void reference(const std::vector<int>& val,
               std::vector<double>& bin) {
  for (const auto v : val) {
    bin[v] += 1.0;
  }
}

void check(const std::vector<double>& bin_ref,
           const cuda_ptr<double>& bin) {
  const auto size = bin_ref.size();
  for (size_t i = 0; i < size; i++) {
    if (bin_ref[i] != bin[i]) {
      std::cout << "mismatch\n";
      std::cout << bin_ref[i] << " " << bin[i] << std::endl;
      std::exit(1);
    }
  }
}

#define BENCH(repr, max, size)                                          \
  do {                                                                  \
    using namespace std::chrono;                                        \
    const int LOOP = 100;                                               \
    const auto beg = system_clock::now();                               \
    for (int i = 0; i < LOOP; i++) repr;                                \
    const auto end = system_clock::now();                               \
    const double dur = duration_cast<milliseconds>(end - beg).count();  \
    std::cerr << "range [0 : " << max << "] ";                          \
    std::cerr << "array size " << size << " ";                          \
    std::cerr << dur << " [ms]\n";                                       \
  } while(0)


#define BENCH_CUDA(repr, max, size, gl_size, tb_size, ...)              \
  do {                                                                  \
    using namespace std::chrono;                                        \
    const int LOOP = 100;                                               \
    const auto beg = system_clock::now();                               \
    for (int i = 0; i < LOOP; i++)                                      \
      repr<<<gl_size, tb_size>>>(__VA_ARGS__) ;                         \
    checkCudaErrors(hipDeviceSynchronize());                           \
    const auto end = system_clock::now();                               \
    const double dur = duration_cast<milliseconds>(end - beg).count();  \
    std::cerr << "range [0 : " << max << "] ";                          \
    std::cerr << "array size " << size << " ";                          \
    std::cerr << dur << " [ms]\n";                                       \
  } while (0)

int main(const int argc, const char* argv[]) {
  int val_size = 10000000;
  int bin_size = 1000;
  if (argc >= 2) val_size   = std::atoi(argv[1]);
  if (argc >= 3) bin_size   = std::atoi(argv[2]);

  cuda_ptr<int> val;
  cuda_ptr<double> bin;

  val.allocate(val_size);
  bin.allocate(bin_size);

  std::mt19937 mt;
  std::uniform_int_distribution<> uid(0, bin_size - 1);
  std::generate_n(&val[0], val_size, [&mt, &uid](){return uid(mt);});
  std::fill_n(&bin[0], bin_size, 0.0);

  val.host2dev();
  bin.host2dev();

  std::vector<int> val_ref(val_size);
  std::vector<double> bin_ref(bin_size);

  std::copy_n(&val[0], val_size, val_ref.begin());
  BENCH(reference(val_ref, bin_ref), bin_size, val_size);

  const auto tb_size = 128;
  const auto gl_size = (val_size - 1) / tb_size + 1;
  BENCH_CUDA(make_hist, bin_size, val_size, gl_size, tb_size, val, bin, val_size);

  bin.dev2host();
  check(bin_ref, bin);
}
