#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <random>
#include <chrono>
#include "cuda_ptr.cuh"

__global__ void make_hist(const int* val,
                          double* bin,
                          const int val_size) {
  const auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < val_size) {
    const auto dst = val[tid];
    atomicAdd(&bin[dst], 1.0);
  }
}

void reference(const std::vector<int>& val,
               std::vector<double>& bin) {
  for (const auto v : val) {
    bin[v] += 1.0;
  }
}

void check(const std::vector<double>& bin_ref,
           const cuda_ptr<double>& bin) {
  const auto size = bin_ref.size();
  for (size_t i = 0; i < size; i++) {
    if (bin_ref[i] != bin[i]) {
      std::cout << "mismatch\n";
      std::cout << bin_ref[i] << " " << bin[i] << std::endl;
      std::exit(1);
    }
  }
}

#define BENCH(repr)                                                     \
  do {                                                                  \
    const auto beg = std::chrono::system_clock::now();                  \
    repr;                                                               \
    const auto end = std::chrono::system_clock::now();                  \
    std::cerr <<                                                        \
      #repr << " " <<                                                   \
      std::chrono::duration_cast<std::chrono::microseconds>(end - beg).count() << \
      " [microsec]\n";                                                  \
  } while(0)

#define BENCH_CUDA(repr, gl_size, tb_size, ...)                         \
  do {                                                                  \
    const auto beg = std::chrono::system_clock::now();                  \
    repr<<<gl_size, tb_size>>>(__VA_ARGS__) ;                           \
    checkCudaErrors(hipDeviceSynchronize());                           \
    const auto end = std::chrono::system_clock::now();                  \
    std::cerr <<                                                        \
      #repr << " " <<                                                   \
      std::chrono::duration_cast<std::chrono::microseconds>(end - beg).count() << \
      " [microsec]\n";                                                  \
  } while(0)

int main(const int argc, const char* argv[]) {
  int val_size = 10000000;
  const int bin_size = 1000;
  if (argc == 2) {
    val_size = std::atoi(argv[1]);
    std::cout << "array size is set to " << val_size << std::endl;
  } else {
    std::cout << "array size is default " << val_size << std::endl;
  }

  cuda_ptr<int> val;
  cuda_ptr<double> bin;

  val.allocate(val_size);
  bin.allocate(bin_size);

  std::mt19937 mt;
  std::uniform_int_distribution<> uid(0, bin_size - 1);
  std::generate_n(&val[0], val_size, [&mt, &uid](){return uid(mt);});
  std::fill_n(&bin[0], bin_size, 0.0);

  val.host2dev();
  bin.host2dev();

  std::vector<int> val_ref(val_size);
  std::vector<double> bin_ref(bin_size);

  std::copy_n(&val[0], val_size, val_ref.begin());
  BENCH(reference(val_ref, bin_ref));

  const auto tb_size = 128;
  const auto gl_size = (val_size - 1) / tb_size + 1;
  BENCH_CUDA(make_hist, gl_size, tb_size, val, bin, val_size);

  bin.dev2host();
  check(bin_ref, bin);
}
