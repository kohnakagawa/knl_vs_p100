#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <random>
#include <chrono>
#include "cuda_ptr.cuh"

__global__ void daxpy(const double* __restrict__ x,
                      const double* __restrict__ y,
                      double* __restrict__ z,
                      const double s,
                      const int val_size) {
  const auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < val_size) {
    z[tid] = s * x[tid] + y[tid];
  }
}

__global__ void daxpy2(const double2* __restrict__ x,
                       const double2* __restrict__ y,
                       double2* __restrict__ z,
                       const double s,
                       const int val_size) {
  const auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < val_size) {
    z[tid].x = s * x[tid].x + y[tid].x;
    z[tid].y = s * x[tid].y + y[tid].y;
  }
}

void reference(const std::vector<double>& x,
               const std::vector<double>& y,
               std::vector<double>& z,
               const double s) {
  const int size = x.size();
  for (int i = 0; i < size; i++) {
    z[i] = s * x[i] + y[i];
  }
}

void check(const std::vector<double>& z_ref,
           const double* z) {
  const auto size = z_ref.size();
  for (size_t i = 0; i < size; i++) {
    if (z_ref[i] != z[i]) {
      std::cout << "mismatch\n";
      std::cout << i << " " << z_ref[i] << " " << z[i] << std::endl;
      std::exit(1);
    }
  }
}

#define BENCH(repr, size)                                               \
  do {                                                                  \
    const auto beg = std::chrono::system_clock::now();                  \
    repr;                                                               \
    const auto end = std::chrono::system_clock::now();                  \
    std::cerr << "array " <<                                            \
      size << " " <<                                                    \
      std::chrono::duration_cast<std::chrono::microseconds>(end - beg).count() << \
      " [microsec]\n";                                                  \
  } while(0)

#define BENCH_CUDA(repr, size, gl_size, tb_size, ...)                   \
  do {                                                                  \
    const auto beg = std::chrono::system_clock::now();                  \
    repr<<<gl_size, tb_size>>>(__VA_ARGS__) ;                           \
    checkCudaErrors(hipDeviceSynchronize());                           \
    const auto end = std::chrono::system_clock::now();                  \
    std::cerr << "array " <<                                            \
      size << " " <<                                                   \
      std::chrono::duration_cast<std::chrono::microseconds>(end - beg).count() << \
      " [microsec]\n";                                                  \
  } while(0)

int main(const int argc, const char* argv[]) {
  int val_size = 1 << 25;
  if (argc == 2) {
    val_size = std::atoi(argv[1]);
  }

  const double s = 2.0;

  cuda_ptr<double> x_vec, y_vec, z_vec;
  cuda_ptr<double2> x2_vec, y2_vec, z2_vec;

  x_vec.allocate(val_size); x2_vec.allocate(val_size / 2);
  y_vec.allocate(val_size); y2_vec.allocate(val_size / 2);
  z_vec.allocate(val_size); z2_vec.allocate(val_size / 2);

  std::mt19937 mt;
  std::uniform_real_distribution<double> urd(0, 1.0);
  std::generate_n(&x_vec[0], val_size, [&mt, &urd](){return urd(mt);});
  std::generate_n(&y_vec[0], val_size, [&mt, &urd](){return urd(mt);});
  std::fill_n(&z_vec[0], val_size, 0.0);

  for (int i = 0; i < val_size; i += 2) {
    x2_vec[i / 2].x = x_vec[i    ];
    x2_vec[i / 2].y = x_vec[i + 1];
    y2_vec[i / 2].x = y_vec[i    ];
    y2_vec[i / 2].y = y_vec[i + 1];
  }

  x_vec.host2dev(); x2_vec.host2dev();
  y_vec.host2dev(); y2_vec.host2dev();
  z_vec.host2dev(); z2_vec.host2dev();

  std::vector<double> x_vec_ref(val_size), y_vec_ref(val_size), z_vec_ref(val_size);

  std::copy_n(&x_vec[0], val_size, x_vec_ref.begin());
  std::copy_n(&y_vec[0], val_size, y_vec_ref.begin());
  std::copy_n(&z_vec[0], val_size, z_vec_ref.begin());
  reference(x_vec_ref, y_vec_ref, z_vec_ref, s);

  const auto tb_size = 128;
  auto gl_size = (val_size - 1) / tb_size + 1;
  // BENCH_CUDA(daxpy, val_size, gl_size, tb_size, x_vec, y_vec, z_vec, s, val_size);
  // z_vec.dev2host();
  // check(z_vec_ref, &z_vec[0]);

  gl_size = (val_size / 2 - 1) / tb_size + 1;
  BENCH_CUDA(daxpy2, val_size, gl_size, tb_size, x2_vec, y2_vec, z2_vec, s, val_size / 2);
  z2_vec.dev2host();
  check(z_vec_ref, &z2_vec[0].x);
}
