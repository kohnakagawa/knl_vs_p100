#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <random>
#include <chrono>
#include "cuda_ptr.cuh"

__global__ void saxpy(const double* __restrict__ x,
                      const double* __restrict__ y,
                      double* __restrict__ z,
                      const double s,
                      const int val_size) {
  const auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < val_size) {
    z[tid] = s * x[tid] + y[tid];
  }
}

void reference(const std::vector<double>& x,
               const std::vector<double>& y,
               std::vector<double>& z,
               const double s) {
  const int size = x.size();
  for (int i = 0; i < size; i++) {
    z[i] = s * x[i] + y[i];
  }
}

void check(const std::vector<double>& z_ref,
           const cuda_ptr<double>& z) {
  const auto size = z_ref.size();
  for (size_t i = 0; i < size; i++) {
    if (z_ref[i] != z[i]) {
      std::cout << "mismatch\n";
      std::cout << z_ref[i] << " " << z[i] << std::endl;
      std::exit(1);
    }
 }
}

#define BENCH(repr)                                                     \
  do {                                                                  \
    const auto beg = std::chrono::system_clock::now();                  \
    repr;                                                               \
    const auto end = std::chrono::system_clock::now();                  \
    std::cerr <<                                                        \
      #repr << " " <<                                                   \
      std::chrono::duration_cast<std::chrono::microseconds>(end - beg).count() << \
      " [microsec]\n";                                                  \
  } while(0)

#define BENCH_CUDA(repr, gl_size, tb_size, ...)                         \
  do {                                                                  \
    const auto beg = std::chrono::system_clock::now();                  \
    repr<<<gl_size, tb_size>>>(__VA_ARGS__) ;                           \
    checkCudaErrors(hipDeviceSynchronize());                           \
    const auto end = std::chrono::system_clock::now();                  \
    std::cerr <<                                                        \
      #repr << " " <<                                                   \
      std::chrono::duration_cast<std::chrono::microseconds>(end - beg).count() << \
      " [microsec]\n";                                                  \
  } while(0)

int main(const int argc, const char* argv[]) {
  int val_size = 10000000;
  if (argc == 2) {
    val_size = std::atoi(argv[1]);
    std::cout << "array size is set to " << val_size << std::endl;
  } else {
    std::cout << "array size is default " << val_size << std::endl;
  }

  const double s = 2.0;

  cuda_ptr<double> x_vec, y_vec, z_vec;

  x_vec.allocate(val_size);
  y_vec.allocate(val_size);
  z_vec.allocate(val_size);

  std::mt19937 mt;
  std::uniform_real_distribution<double> urd(0, 1.0);
  std::generate_n(&x_vec[0], val_size, [&mt, &urd](){return urd(mt);});
  std::generate_n(&y_vec[0], val_size, [&mt, &urd](){return urd(mt);});
  std::fill_n(&z_vec[0], val_size, 0.0);

  x_vec.host2dev();
  y_vec.host2dev();
  z_vec.host2dev();

  std::vector<double> x_vec_ref(val_size), y_vec_ref(val_size), z_vec_ref(val_size);

  std::copy_n(&x_vec[0], val_size, x_vec_ref.begin());
  std::copy_n(&y_vec[0], val_size, y_vec_ref.begin());
  std::copy_n(&z_vec[0], val_size, z_vec_ref.begin());
  BENCH(reference(x_vec_ref, y_vec_ref, z_vec_ref, s));

  const auto tb_size = 128;
  const auto gl_size = (val_size - 1) / tb_size + 1;
  BENCH_CUDA(saxpy, gl_size, tb_size, x_vec, y_vec, z_vec, s, val_size);

  z_vec.dev2host();
  check(z_vec_ref, z_vec);
}
