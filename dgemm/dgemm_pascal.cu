#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <random>
#include <chrono>
#include <hipblas.h>
#include <cassert>
#include "../cuda_ptr.cuh"

static inline void __cublasSafeCall(hipblasStatus_t err,
                                    const char *file,
                                    const int line) {
  if (HIPBLAS_STATUS_SUCCESS != err) {
    fprintf(stderr, "CUBLAS error in file '%s', line %d\n \nerror %d \nterminating!\n", file, line, err);
    hipDeviceReset();
    assert(0);
  }
}

#ifndef cublasSafeCall
#define cublasSafeCall(err)     __cublasSafeCall(err, __FILE__, __LINE__)
#endif

constexpr int mat_size = 1024;
constexpr int mat_block_size = 128;
static_assert(mat_size % mat_block_size == 0, "error");

__device__ __forceinline__ int lane_id() {
  return threadIdx.x % warpSize;
}

template <typename T>
__device__ __forceinline__ T warp_segment_reduce(T var) {
  for (int offset = (warpSize >> 1); offset > 0; offset >>= 1) {
    var += __shfl_down(var, offset);
  }
  return var;
}

// ASSUME: mat_block_size == blockDim.x
__global__ void dgemm(const double* __restrict__ x,
                      const double* __restrict__ y,
                      double* __restrict__ z,
                      const int N = mat_size) {
  const auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < mat_size * mat_size) {
    const auto i = tid / N;
    const auto j = tid % N;
    double sum = 0.0;
    for (int k = 0; k < mat_size; k++) {
      sum += x[N * i + k] * y[N * k + j];
    }
    z[tid] = sum;
  }
}

void dgemm_cublas(const cuda_ptr<double>& x,
                  const cuda_ptr<double>& y,
                  cuda_ptr<double>& z,
                  const hipblasHandle_t& handle) {
  const double alpha = 1.0, beta = 1.0;
  cublasSafeCall(hipblasDgemm(handle,
                             HIPBLAS_OP_T, HIPBLAS_OP_T,
                             mat_size, mat_size, mat_size,
                             &alpha,
                             x, mat_size,
                             y, mat_size,
                             &beta,
                             z, mat_size));
}

void reference(const double* __restrict x,
               const double* __restrict y,
               double* __restrict z,
               const int N = mat_size) {
#if 0
#pragma omp parallel for
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      for (int k = 0; k < N; k++) {
        z[N * i + j] += x[N * i + k] * y[N * k + j];
      }
    }
  }
#else
  const int nib = 4;
  const int njb = 4;
  const int nkb = 4;

#pragma omp parallel for
  for (int ib = 0; ib < N; ib += nib)
    for (int jb = 0; jb < N; jb += njb)
      for (int kb = 0; kb < N; kb += nkb)
        for (int i = ib; i < ib + nib; i++)
          for (int j = jb; j < jb + njb; j++)
            for (int k = kb; k < kb + nkb; k++)
              z[N * i + j] += x[N * i + k] * y[N * k + j];
#endif
}

void check_cublas(const double* z_ref,
                  const double* z,
                  const double eps = 1.0e-8) {
  for (int i = 0; i < mat_size; i++) {
    for (int j = 0; j < mat_size; j++) {
      if (std::abs(z_ref[mat_size * j + i] - z[mat_size * i + j]) >= eps) {
        std::cout << "mismatch\n";
        std::cout << i << " " << j << " "
                  << z_ref[mat_size * j + i] << " "
                  << z[mat_size * i + j] << std::endl;
        std::exit(1);
      }
    }
  }
}

void check(const double* z_ref,
           const double* z,
           const double eps = 1.0e-8) {
  for (int i = 0; i < mat_size * mat_size; i++) {
    if (std::abs(z_ref[i] - z[i] >= eps)) {
      std::cout << "mismatch\n";
      std::cout << i << " " << z_ref[i] << " " << z[i] << std::endl;
      std::exit(1);
    }
  }
}

void show_version(const hipblasHandle_t& handle) {
  int cublas_version = 0;
  cublasSafeCall(cublasGetVersion(handle, &cublas_version));
  std::cout << "cublas version " << cublas_version << "\n";
}

#define BENCH(repr)                                                     \
  do {                                                                  \
    using namespace std::chrono;                                        \
    const auto beg = system_clock::now();                               \
    repr;                                                               \
    const auto end = system_clock::now();                               \
    const double elapsed =                                              \
      1.0e-3 * duration_cast<milliseconds>(end - beg).count();          \
    const double flops =                                                \
      2.0 * 1.0e-9 * mat_size * mat_size * mat_size / elapsed;          \
    std::cerr << "array " << mat_size << " " << flops << " [GFLOPS] ";  \
    std::cerr << elapsed << " [ms]\n";                                  \
  } while (0)

#define BENCH_CUDA(repr, gr_size, tb_size, ...)                         \
  do {                                                                  \
    using namespace std::chrono;                                        \
    const auto beg = system_clock::now();                               \
    repr<<<gr_size, tb_size>>>(__VA_ARGS__) ;                           \
    checkCudaErrors(hipDeviceSynchronize());                           \
    const auto end = system_clock::now();                               \
    const double elapsed =                                              \
      1.0e-3 * duration_cast<milliseconds>(end - beg).count();          \
    const double flops =                                                \
      2.0 * 1.0e-9 * mat_size * mat_size * mat_size / elapsed;          \
    std::cerr << "array " << mat_size << " " << flops << " [GFLOPS] ";  \
    std::cerr << elapsed << " [ms]\n";                                  \
  } while (0)

int main() {
  cuda_ptr<double> x_mat, y_mat, z_mat;
  cuda_ptr<double> x_mat_bl, y_mat_bl, z_mat_bl;

  x_mat.allocate(mat_size * mat_size);
  y_mat.allocate(mat_size * mat_size);
  z_mat.allocate(mat_size * mat_size);

  std::mt19937 mt;
  std::uniform_real_distribution<double> urd(0, 1.0);
  std::generate_n(&x_mat[0], mat_size * mat_size, [&mt, &urd](){return urd(mt);});
  std::generate_n(&y_mat[0], mat_size * mat_size, [&mt, &urd](){return urd(mt);});
  std::fill_n(&z_mat[0], mat_size * mat_size, 0.0);

  x_mat.host2dev();
  y_mat.host2dev();
  z_mat.host2dev();

  double *x_mat_ref = new double [mat_size * mat_size];
  double *y_mat_ref = new double [mat_size * mat_size];
  double *z_mat_ref = new double [mat_size * mat_size];

  std::copy_n(&x_mat[0], mat_size * mat_size, x_mat_ref);
  std::copy_n(&y_mat[0], mat_size * mat_size, y_mat_ref);
  std::copy_n(&z_mat[0], mat_size * mat_size, z_mat_ref);
  BENCH(reference(x_mat_ref, y_mat_ref, z_mat_ref));

#if 0
  const auto tot_threads = mat_size * mat_size;
  const auto tb_size = 128;
  const auto gr_size = (tot_threads - 1) / tb_size + 1;
  BENCH_CUDA(dgemm, gr_size, tb_size, x_mat, y_mat, z_mat);
  z_mat.dev2host();
  check(z_mat_ref, &z_mat[0]);
#else
  hipblasHandle_t handle;
  cublasSafeCall(hipblasCreate(&handle));
  show_version(handle);
  BENCH(dgemm_cublas(x_mat, y_mat, z_mat, handle));
  cublasSafeCall(hipblasDestroy(handle));
  z_mat.dev2host();
  check_cublas(z_mat_ref, &z_mat[0]);
#endif

  delete [] x_mat_ref;
  delete [] y_mat_ref;
  delete [] z_mat_ref;
}
